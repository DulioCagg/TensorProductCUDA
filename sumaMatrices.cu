
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define M 10
#define N 10

// __device__ double z[128][128];

double a[M][N], b[M][N], c[M][N];

//Hace toda la poop.
__global__ void kernelSumaMatrices(double *a, double *b, double *c, int m, int n)
{
int i = threadIdx.x;
int j = threadIdx.y;

//   a[i][j] = b[i][j] + b[i][j];
   a[i*n+j] = b[i*n+j] + c[i*n+j];
}

void sumaMatricesEnDevice(double *a, double *b, double *c, int m, int n) {
 double *aD, *bD, *cD;
 int size=m*n*sizeof(double);
 dim3 bloques(1,1);
 dim3 hilos(10,10);

// 1. Asignar memoria
  hipMalloc(&aD, size);
  hipMalloc(&bD, size);
  hipMalloc(&cD, size);

  hipSetDevice(0);
  hipMalloc(&cD, size);

// 2. Copiar datos del Host al Device
   hipMemcpy(bD, b, size, hipMemcpyHostToDevice);
   hipMemcpy(cD, c, size, hipMemcpyHostToDevice);
//   cudaMemcpy(aD, a, size, cudaMemcpyDefault);
//   cudaMemcpy(bD, b, size, cudaMemcpyDefault);

   // 3. Ejecutar kernel
   kernelSumaMatrices<<<bloques , hilos>>>(aD, bD, cD, m, n);

// 4. Copiar datos del device al Host
  hipMemcpy(a, aD, size, hipMemcpyDeviceToHost);
  //cudaMemcpy(a, aD, size, cudaMemcpyDefault);

// 5. Liberar Memoria
hipFree(aD); hipFree(bD); hipFree(cD);
}

int main() {
int i, j;

//   a=(double *)malloc(M*N*sizeof(double));
//   b=(double *)malloc(M*N*sizeof(double));
//   c=(double *)malloc(M*N*sizeof(double));

  for (i=0; i<M; i++) {
     for (j=0; j<N; j++) {
        b[i][j] = c[i][j] = i+j;
     }
  }

sumaMatricesEnDevice((double *)a, (double *)b, (double *)c, M, N);

  for (i=0; i<M; i++) {
     for (j=0; j<N; j++) {
        printf("%3.2f ", a[i][j]);
     }
     printf("\n");
  }

  free(a); free(b); free(c);
}

