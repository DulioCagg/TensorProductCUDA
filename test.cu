
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N 2
#define M 3

int a[N][M], b[N][M], c[N*N][M*M];

__global__ void tensorProduct(int rB, int cB, int *a, int *b, int *result)
{
  int i = threadIdx.x;
  int j = threadIdx.y;  


  //Se queda.
  for (int k = 0; k < rB; k++)
  {
    for (int l = 0; l < cB; l++)
    {
      result[(i * rB + k) * cB * cB + (j * cB + l)] = a[i*cB+j] * b[k*cB+l];
    }
  }
}

void printMatrix(int matrix[N][M])
{
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < M; j++)
    {
      printf("%d\t", matrix[i][j]);
    }
    printf("\n");
  }
}

void initMatrix(int matrix[N][M])
{
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < M; j++)
    {
      matrix[i][j] = rand() % 5;
    }
  }
}

void tensorProductDevice(int rB, int cB, int *a, int *b, int *c){
  int *aD, *bD, *cD;
  int size = rB * cB * sizeof(int);
  int sizeRes = rB * rB * cB * cB * sizeof(int);

  dim3 bloques(1,1);
  dim3 hilos(N,M);

  hipMalloc(&aD, size);
  hipMalloc(&bD, size);
  hipMalloc(&cD, sizeRes);

  hipSetDevice(0);
  hipMemcpy(aD, a, size, hipMemcpyHostToDevice);
  hipMemcpy(bD, b, size, hipMemcpyHostToDevice);

  tensorProduct<<<bloques , hilos>>>(rB, cB, aD, bD, cD);
  
  hipMemcpy(c, cD, sizeRes, hipMemcpyDeviceToHost);
  
  hipFree(aD);
  hipFree(bD);
  hipFree(cD);

}

int main()
{
  //int rA = 2;
  //int cA = 2;
  //int cB = 2;
  //int rB = 3;
  //int rR = rA * rB;
  //int cR = cA * cB;

  srand(time(NULL));
  initMatrix(a);
  initMatrix(b);

  tensorProductDevice(N, M, (int *) a, (int *) b, (int *) c);

  printf("Matrix A:\n");
  printMatrix(a);

  printf("Matrix B:\n");
  printMatrix(b);

  printf("Result: R:\n");
  for (int i = 0; i < N * N; i++)
  {
    for (int j = 0; j < M * M; j++)
    {
      printf("%d\t", c[i][j]);
    }
    printf("\n");
  }
  
  free(a);
  free(b);
  free(c);
}